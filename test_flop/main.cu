
#include <hip/hip_runtime.h>
#include <iostream>
#include <string.h>

void __global__ run(float * h)
{
	int idx = blockIdx.x*64+threadIdx.x;
	if (idx > 10000) return;
	h[idx] += 1.3f;
}

int main(int argc, char ** argv)
{
	int times = atoi(argv[1]);
	float * h_d;
	hipMalloc(&h_d, 10000*sizeof(float));
	for (int i = 0; i < times; ++i)
		run<<<157, 64>>>(h_d);
	return 0;
}
